#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "fp16_conversion.h"

using namespace std;



const char *cublasGetErrorString(hipblasStatus_t status)
{
  switch (status)
  {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "unknown error";
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
void CPU_fill_rand(hipComplex *A, int nr_rows_A, int nr_cols_A)
{
  int a = 1;

  for (int i = 0; i < nr_rows_A * nr_cols_A; i++)
  {
    A[i] = {(float)rand() / (float)(RAND_MAX / a), 0};
  }
}

int main(int argc, char **argv)
{

  int min_m_k_n = 2;
  int max_m_k_n = 4096 * 4;
  int repeats = 2;
  int verbose = 1;

  cout << "\ncublasCgemm test result:\n"
       << endl;

  if (verbose)
    cout << "running with"
         << " min_m_k_n: " << min_m_k_n
         << " max_m_k_n: " << max_m_k_n
         << " repeats: " << repeats
         << endl;

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  if (verbose)
    cout << "allocating device variables" << endl;

  // Allocate 3 arrays on CPU

  hipComplex *h_A = (hipComplex *)malloc(max_m_k_n * max_m_k_n * sizeof(hipComplex));
  hipComplex *h_B = (hipComplex *)malloc(max_m_k_n * max_m_k_n * sizeof(hipComplex));
  hipComplex *h_C = (hipComplex *)malloc(max_m_k_n * max_m_k_n * sizeof(hipComplex));

  CPU_fill_rand(h_A, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_B, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_C, max_m_k_n, max_m_k_n);

  // Allocate 3 arrays on GPU
  hipComplex *d_A, *d_B, *d_C;
  checkCuda(hipMallocManaged(&d_A, max_m_k_n * max_m_k_n * sizeof(hipComplex)));
  checkCuda(hipMallocManaged(&d_B, max_m_k_n * max_m_k_n * sizeof(hipComplex)));
  checkCuda(hipMallocManaged(&d_C, max_m_k_n * max_m_k_n * sizeof(hipComplex)));

  checkCuda(hipMemcpy(d_A, h_A, max_m_k_n * max_m_k_n * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_B, h_B, max_m_k_n * max_m_k_n * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_C, h_C, max_m_k_n * max_m_k_n * sizeof(hipComplex), hipMemcpyHostToDevice));

  int lda, ldb, ldc, m, n, k;
  const hipComplex alf = {1.0f, 0};
  const hipComplex bet = {0.0f, 0};
  const hipComplex *alpha = &alf;
  const hipComplex *beta = &bet;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int size = min_m_k_n; size <= max_m_k_n; size = size * 2)
  {
    double sum = 0.0;
    for (int rep = 0; rep < repeats; rep++)
    {
      hipEventRecord(start, 0);
      m = n = k = size;
      lda = m;
      ldb = k;
      ldc = m;

      stat = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc);

      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      if (stat != HIPBLAS_STATUS_SUCCESS)
      {
        cerr << "hipblasSgemmBatched failed" << endl;
        exit(1);
      }
      assert(!hipGetLastError());

      float elapsed;
      hipEventElapsedTime(&elapsed, start, stop);
      elapsed /= 1000.0f;
      sum += elapsed;
    }

    cout << "complex32: size "

         << size << " average: " << sum / repeats << " s " << endl;
  }

  // Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
